#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mpi.h>

#include "../tools/utils.h"
#include "GlobalGPU.cuh"

// Cuda Error checking macro
#define CHECK_CUDA_ERROR(call)                                                 \
    do                                                                         \
    {                                                                          \
        hipError_t error = call;                                              \
        if (error != hipSuccess)                                              \
        {                                                                      \
            std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " \
                      << __FILE__ << ":" << __LINE__ << std::endl;             \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

__global__ void assignPointsToGlobalCentroids(float *data, float *centroids, int *assignments, int numPoints, int numDimensions, int k)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numPoints)
    {
        float minDist = __FLT_MAX__;
        int nearestCentroid = 0;

        for (int i = 0; i < k; ++i)
        {
            float distance = 0.0f;
            for (int j = 0; j < numDimensions; ++j)
            {
                float diff = data[idx * numDimensions + j] - centroids[i * numDimensions + j];
                distance += diff * diff;
            }

            if (distance < minDist)
            {
                minDist = distance;
                nearestCentroid = i;
            }
        }

        assignments[idx] = nearestCentroid;
    }
}

// CUDA Kernel for accumulating points for centroid updates
__global__ void updateCentroidsGlobalKernel(float *data, float *centroids, int *assignments, int *clusterCounts, int numPoints, int numDimensions, int k)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numPoints)
    {
        int clusterId = assignments[idx];

        // Atomic operation to accumulate the sum of points for each centroid
        for (int j = 0; j < numDimensions; ++j)
        {
            atomicAdd(&centroids[clusterId * numDimensions + j], data[idx * numDimensions + j]);
        }

        atomicAdd(&clusterCounts[clusterId], 1);
    }
}

// CUDA Kernel for normalzing centroids by cluster size
__global__ void normalizeCentroidsGlobalKernel(float *centroids, int *clusterCounts, int k, int numDimensions)
{
    int clusterId = blockIdx.x * blockDim.x + threadIdx.x;

    if (clusterId < k && clusterCounts[clusterId] > 0)
    {
        for (int j = 0; j < numDimensions; ++j)
        {
            centroids[clusterId * numDimensions + j] /= clusterCounts[clusterId];
        }
    }
}

SpotifyGenreRevealParty::GlobalGPU::GlobalGPU(int clusters, int maxIterations)
    : m_number_of_clusters(clusters),
      m_max_iterations(maxIterations),
      m_rank(0),
      m_num_processes(0),
      m_num_points_per_process(0),
      m_cuda_aware_mpi(false),
      m_device_data(nullptr),
      m_device_centroids(nullptr),
      m_device_temp_centroids(nullptr),
      m_device_cluster_assignments(nullptr),
      m_device_cluster_count(nullptr)
{
    initializeMPI();
}

SpotifyGenreRevealParty::GlobalGPU::~GlobalGPU()
{
    this->freeGPUMemory();
}

void SpotifyGenreRevealParty::GlobalGPU::initializeMPI()
{
    // Initialize MPI
    int initialized;
    MPI_Initialized(&initialized);

    if (!initialized)
    {
        // Only print this warning from process 0
        int rank = 0;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        if (rank == 0)
        {
            std::cerr << "WARNING: MPI was not initialized by main(). "
                      << "This may cause issues with proper MPI cleanup." << std::endl;
        }

        // Don't try to initialize MPI here - it should be done in main
        // instead, throw an error
        throw std::runtime_error("MPI should be initialized in main before creating algorithms");
    }

    // Get rank and size
    MPI_Comm_rank(MPI_COMM_WORLD, &this->m_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &this->m_num_processes);

    // select gpu based on local rank
    char *local_rank_str = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
    int local_rank = (local_rank_str != nullptr) ? atoi(local_rank_str) : this->m_rank;

    // Get number of available gpus
    int deviceCount;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));

    if (deviceCount > 0)
    {
        int device = local_rank % deviceCount;
        CHECK_CUDA_ERROR(hipSetDevice(device));

        if (this->m_rank == 0)
        {
            std::cout << "Using GPU device " << device << " on rank " << this->m_rank << std::endl;
        }
    }
    else
    {
        throw std::runtime_error("No GPU devices available");
    }

    this->m_cuda_aware_mpi = this->isMpiCudaAware();

    if (this->m_rank == 0)
    {
        std::cout << "CUDA-aware MPI: " << (this->m_cuda_aware_mpi ? "Enabled" : "Disabled") << std::endl;
    }
}

void SpotifyGenreRevealParty::GlobalGPU::freeGPUMemory()
{
    // Free all device memory
    if (this->m_device_data)
        CHECK_CUDA_ERROR(hipFree(this->m_device_data));
    if (this->m_device_centroids)
        CHECK_CUDA_ERROR(hipFree(this->m_device_centroids));
    if (this->m_device_temp_centroids)
        CHECK_CUDA_ERROR(hipFree(this->m_device_temp_centroids));
    if (this->m_device_cluster_assignments)
        CHECK_CUDA_ERROR(hipFree(this->m_device_cluster_assignments));
    if (this->m_device_cluster_count)
        CHECK_CUDA_ERROR(hipFree(this->m_device_cluster_count));

    this->m_device_data = nullptr;
    this->m_device_centroids = nullptr;
    this->m_device_temp_centroids = nullptr;
    this->m_device_cluster_assignments = nullptr;
    this->m_device_cluster_count = nullptr;
}

bool SpotifyGenreRevealParty::GlobalGPU::isMpiCudaAware()
{
#ifdef MPIX_CUDA_AWARE_SUPPORT
    if (MPIX_CUDA_AWARE_SUPPORT)
    {
        return true;
    }
    else
    {
        return false;
    }
#elif defined(MVAPICH2_NUMVERSION) && (MVAPICH2_NUMVERSION >= 20000000)
    // MVAPICH2 2.0+ supports CUDA
    return true;
#elif defined(OPEN_MPI) && (OPEN_MPI >= 1005004)
    // OpenMPI 1.5.4+ might support CUDA
    char *env_var = getenv("OMPI_MCA_mpi_cuda_support");
    if (env_var && strcmp(env_var, "1") == 0)
    {
        return true;
    }
    return false;
#else
    // Default to false to be safe
    if (m_rank == 0)
    {
        std::cout << "CUDA-aware MPI detection not available, defaulting to CPU buffers" << std::endl;
    }
    return false;
#endif
}

void SpotifyGenreRevealParty::GlobalGPU::allocateGPUMemory()
{
    // Calculate memory requirements
    std::size_t data_size = this->m_num_points_per_process * this->m_number_of_dimensions * sizeof(float);
    std::size_t centroid_size = this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float);
    std::size_t assignment_size = this->m_num_points_per_process * sizeof(int);
    std::size_t count_size = this->m_number_of_clusters * sizeof(int);

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc(&this->m_device_data, data_size));
    CHECK_CUDA_ERROR(hipMalloc(&this->m_device_centroids, centroid_size));
    CHECK_CUDA_ERROR(hipMalloc(&this->m_device_temp_centroids, centroid_size));
    CHECK_CUDA_ERROR(hipMalloc(&this->m_device_cluster_assignments, assignment_size));
    CHECK_CUDA_ERROR(hipMalloc(&this->m_device_cluster_count, count_size));

    // Initialize assignments and counts to zero
    CHECK_CUDA_ERROR(hipMemset(this->m_device_cluster_assignments, 0, assignment_size));
    CHECK_CUDA_ERROR(hipMemset(this->m_device_cluster_count, 0, count_size));

    // Alocate host buffers
    this->m_host_cluster_assignments.resize(this->m_num_points_per_process, 0);

    if (this->m_rank == 0)
    {
        std::cout << "Allocated GPU memory: "
                  << (data_size + 2 * centroid_size + assignment_size + count_size) / (1024 * 1024)
                  << "MB on rank " << m_rank << std::endl;
    }
}

void SpotifyGenreRevealParty::GlobalGPU::distributeData(const std::vector<SpotifyGenreRevealParty::Point> &data)
{
    int total_points = static_cast<int>(data.size());

    // Calculate number of points per process
    this->m_num_points_per_process = total_points / this->m_num_processes;
    int remainder = total_points % this->m_num_processes;

    // Last process gets any remainder poitns
    if (this->m_rank == this->m_num_processes - 1)
    {
        this->m_num_points_per_process += remainder;
    }

    // Calculate start index for this process' data
    int start_idx = this->m_rank * (total_points / this->m_num_processes);

    this->m_host_flat_data.resize(this->m_num_points_per_process * this->m_number_of_dimensions);

    if (this->m_rank == 0)
    {
        // process 0 has the original data, extract its portion
        for (int i = 0; i < this->m_num_points_per_process; ++i)
        {
            for (int j = 0; j < this->m_number_of_dimensions; ++j)
            {
                this->m_host_flat_data[i * this->m_number_of_dimensions + j] = data[start_idx + i].features[j];
            }
        }

        // Send data to other processes
        for (int proc = 1; proc < this->m_num_processes; ++proc)
        {
            int proc_start_idx = proc * (total_points / this->m_num_processes);
            int proc_num_points = (proc == this->m_num_processes - 1) ? (total_points / this->m_num_processes) + remainder : (total_points / this->m_num_processes);

            // Last process gets any remainder points
            std::vector<float> proc_data(proc_num_points * this->m_number_of_dimensions);

            // Pack data for the process
            for (int i = 0; i < proc_num_points; ++i)
            {
                for (int j = 0; j < this->m_number_of_dimensions; ++j)
                {
                    proc_data[i * this->m_number_of_dimensions + j] = data[proc_start_idx + i].features[j];
                }
            }

            // Send data to the process
            MPI_Send(proc_data.data(), proc_num_points * this->m_number_of_dimensions, MPI_FLOAT, proc, 0, MPI_COMM_WORLD);
        }
    }
    else
    {
        // Receive data from process 0
        MPI_Recv(this->m_host_flat_data.data(), this->m_num_points_per_process * this->m_number_of_dimensions, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    // Copy data to gpu
    CHECK_CUDA_ERROR(hipMemcpy(this->m_device_data, this->m_host_flat_data.data(), this->m_num_points_per_process * this->m_number_of_dimensions * sizeof(float), hipMemcpyHostToDevice));
}

void SpotifyGenreRevealParty::GlobalGPU::initializeCentroids(int k, int dimensions)
{
    // Resize host centroids vector
    this->m_host_centroids.resize(k * dimensions);

    // process 0 initializes centroids
    if (this->m_rank == 0)
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dist(0, this->m_num_points_per_process - 1);

        // Randomly select k unique points from the data
        for (int i = 0; i < k; ++i)
        {
            int point_idx = dist(gen);
            for (int j = 0; j < dimensions; ++j)
            {
                this->m_host_centroids[i * dimensions + j] = this->m_host_flat_data[point_idx * dimensions + j];
            }
        }
    }

    // Broadcast the centroids to all processes
    MPI_Bcast(this->m_host_centroids.data(), k * dimensions, MPI_FLOAT, 0, MPI_COMM_WORLD);

    // Copy centroids to GPU
    CHECK_CUDA_ERROR(hipMemcpy(this->m_device_centroids, this->m_host_centroids.data(), k * dimensions * sizeof(float), hipMemcpyHostToDevice));
}

void SpotifyGenreRevealParty::GlobalGPU::runDistributedKMeans(double tolerance)
{
    const int threadsPerBlock = 256;
    const int blocksForPoints = (this->m_num_points_per_process + threadsPerBlock - 1) / threadsPerBlock;
    const int blocksForCentroids = (this->m_number_of_clusters + threadsPerBlock - 1) / threadsPerBlock;

    // For convergence check
    std::vector<float> prev_centroids(this->m_number_of_clusters * this->m_number_of_dimensions);

    // Temporary buffers for reduction
    std::vector<float> local_centroids_sums(this->m_number_of_clusters * this->m_number_of_dimensions, 0.0f);
    std::vector<int> local_centroid_counts(this->m_number_of_clusters, 0);
    std::vector<float> global_centroids_sums(this->m_number_of_clusters * this->m_number_of_dimensions, 0.0f);
    std::vector<int> global_centroid_counts(this->m_number_of_clusters, 0);

    // Main KMeans loop
    for (int iter = 0; iter < this->m_max_iterations; ++iter)
    {
        // save current centroids for convergence check
        std::copy(this->m_host_centroids.begin(), this->m_host_centroids.end(), prev_centroids.begin());

        // Reset temporary buffers
        CHECK_CUDA_ERROR(hipMemset(this->m_device_temp_centroids, 0, this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float)));
        CHECK_CUDA_ERROR(hipMemset(this->m_device_cluster_count, 0, this->m_number_of_clusters * sizeof(int)));

        // Step 1: assign points to nearest centroid
        assignPointsToGlobalCentroids<<<blocksForPoints, threadsPerBlock>>>(this->m_device_data, this->m_device_centroids, this->m_device_cluster_assignments, this->m_num_points_per_process, this->m_number_of_dimensions, this->m_number_of_clusters);
        CHECK_CUDA_ERROR(hipGetLastError());

        // Step 2. Update centroids (accumulate sum)
        updateCentroidsGlobalKernel<<<blocksForPoints, threadsPerBlock>>>(this->m_device_data, this->m_device_temp_centroids, this->m_device_cluster_assignments, this->m_device_cluster_count, this->m_num_points_per_process, this->m_number_of_dimensions, this->m_number_of_clusters);
        CHECK_CUDA_ERROR(hipGetLastError());

        // Step 3. Global Reduction
        if (this->m_cuda_aware_mpi)
        {
            // CUDA aware MPI: use GPU memory directly
            MPI_Allreduce(MPI_IN_PLACE, this->m_device_temp_centroids, this->m_number_of_clusters * this->m_number_of_dimensions, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
            MPI_Allreduce(MPI_IN_PLACE, this->m_device_cluster_count, this->m_number_of_clusters, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

            // normalize centroids
            normalizeCentroidsGlobalKernel<<<blocksForCentroids, threadsPerBlock>>>(this->m_device_temp_centroids, this->m_device_cluster_count, this->m_number_of_clusters, this->m_number_of_dimensions);
            CHECK_CUDA_ERROR(hipGetLastError());

            // Copy to main centroids buffer
            CHECK_CUDA_ERROR(hipMemcpy(this->m_device_centroids, this->m_device_temp_centroids, this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float), hipMemcpyDeviceToDevice));
        }
        else
        {
            // Traditional MPI Approach - copy through host
            CHECK_CUDA_ERROR(hipMemcpy(local_centroids_sums.data(), this->m_device_temp_centroids,
                                        this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float),
                                        hipMemcpyDeviceToHost));

            CHECK_CUDA_ERROR(hipMemcpy(local_centroid_counts.data(), this->m_device_cluster_count,
                                        this->m_number_of_clusters * sizeof(int),
                                        hipMemcpyDeviceToHost));

            // Reduce across processes
            MPI_Allreduce(local_centroids_sums.data(), global_centroids_sums.data(),
                          this->m_number_of_clusters * this->m_number_of_dimensions,
                          MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);

            MPI_Allreduce(local_centroid_counts.data(), global_centroid_counts.data(),
                          this->m_number_of_clusters, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

            // Normalize centroids on CPU
            for (int i = 0; i < this->m_number_of_clusters; ++i)
            {
                if (global_centroid_counts[i] > 0)
                {
                    for (int j = 0; j < this->m_number_of_dimensions; ++j)
                    {
                        this->m_host_centroids[i * this->m_number_of_dimensions + j] =
                            global_centroids_sums[i * this->m_number_of_dimensions + j] / global_centroid_counts[i];
                    }
                }
            }

            // Copy updated centroids back to GPU
            CHECK_CUDA_ERROR(hipMemcpy(this->m_device_centroids, this->m_host_centroids.data(),
                                        this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float),
                                        hipMemcpyHostToDevice));
        }

        // Step 4. check for convergence
        bool local_converged = true;

        // copy centroids back to host if using CUDA aware MPI
        if (this->m_cuda_aware_mpi)
        {
            CHECK_CUDA_ERROR(hipMemcpy(this->m_host_centroids.data(), this->m_device_centroids, this->m_number_of_clusters * this->m_number_of_dimensions * sizeof(float), hipMemcpyDeviceToHost));
        }

        // check if centorids have changed significantly
        for (int i = 0; i < this->m_number_of_clusters * this->m_number_of_dimensions; i++)
        {
            if (std::abs(prev_centroids[i] - this->m_host_centroids[i]) > tolerance)
            {
                local_converged = false;
                break;
            }
        }

        // Check if all proceses agree on convergence
        bool global_converged = false;
        MPI_Allreduce(&local_converged, &global_converged, 1, MPI_CXX_BOOL, MPI_LAND, MPI_COMM_WORLD);

        if (global_converged)
        {
            if (this->m_rank == 0)
            {
                std::cout << "Converged after " << iter + 1 << " iterations." << std::endl;
            }
            break;
        }

        // status update every 10 iters
        if (this->m_rank == 0 && (iter + 1) % 10 == 0)
        {
            std::cout << "Iteration " << iter + 1 << " completed." << std::endl;
        }
    }

    // Copy final cluster Assignments to host
    CHECK_CUDA_ERROR(hipMemcpy(this->m_host_cluster_assignments.data(), this->m_device_cluster_assignments, this->m_num_points_per_process * sizeof(int), hipMemcpyDeviceToHost));
}

void SpotifyGenreRevealParty::GlobalGPU::gatherResults(std::vector<SpotifyGenreRevealParty::Point> &data)
{
    // Gather all cluster assignments from all processes
    int total_points = static_cast<int>(data.size());

    // First gather the counts from each process
    std::vector<int> recv_counts(this->m_num_processes);
    std::vector<int> displacements(this->m_num_processes);

    MPI_Gather(&this->m_num_points_per_process, 1, MPI_INT, recv_counts.data(), 1, MPI_INT, 0, MPI_COMM_WORLD);

    // Calculate displacements for each process
    if (this->m_rank == 0)
    {
        this->m_global_cluster_assignments.resize(total_points);
        int displacement = 0;

        for (int i = 0; i < this->m_num_processes; ++i)
        {
            displacements[i] = displacement;
            displacement += recv_counts[i];
        }
    }

    // Gather all assignments to rank 0
    MPI_Gatherv(this->m_host_cluster_assignments.data(), this->m_num_points_per_process, MPI_INT, this->m_rank == 0 ? this->m_global_cluster_assignments.data() : nullptr, recv_counts.data(), displacements.data(), MPI_INT, 0, MPI_COMM_WORLD);

    // Update original dat with cluster assignments (only on rank 0)
    if (this->m_rank == 0)
    {
        for (int i = 0; i < total_points; ++i)
        {
            data[i].clusterId = this->m_global_cluster_assignments[i];
        }

        // prlint cluster stats
        std::vector<int> cluster_sizes(this->m_number_of_clusters, 0);
        for (int i = 0; i < total_points; ++i)
        {
            cluster_sizes[this->m_global_cluster_assignments[i]]++;
        }

        std::cout << "\nClustering results: " << std::endl;
        std::cout << "Total points: " << total_points << std::endl;
        std::cout << "Clusters: " << this->m_number_of_clusters << std::endl;
        std::cout << "Cluster sizes: " << std::endl;
        for (int i = 0; i < this->m_number_of_clusters; ++i)
        {
            std::cout << "Cluster " << i << ": " << cluster_sizes[i] << " points" << std::endl;
        }
        std::cout << "----------------------------------------" << std::endl;
    }
}

void SpotifyGenreRevealParty::GlobalGPU::run(std::vector<SpotifyGenreRevealParty::Point> &data, int k, std::size_t dimensions, int maxIterations, double tolerance)
{
    auto start_time = std::chrono::high_resolution_clock::now();

    std::cout << "Rank " << this->m_rank << ": Starting GlobalGPU run " << std::endl;
    this->m_number_of_dimensions = static_cast<int>(dimensions);
    this->m_number_of_clusters = k;
    this->m_max_iterations = maxIterations;

    // Step 1. Calculate data distribution
    this->calculateDataDistribution(data.size());

    // Step 2: NOW allocate GPU memory
    this->allocateGPUMemory();

    // Step 3: Distribute the actual data
    this->distributeData(data);

    // Step 3: Initialize centroids
    this->initializeCentroids(k, this->m_number_of_dimensions);

    if (this->m_rank == 0)
    {
        std::cout << "Running KMeans with " << this->m_number_of_clusters
                  << " clusters and " << this->m_max_iterations
                  << " max iterations." << std::endl;
    }

    // Step 4: Run distributed KMeans
    this->runDistributedKMeans(tolerance);

    // Step 5: Gather results
    this->gatherResults(data);

    // Step 6: Save results to CSV on rank 0
    if (this->m_rank == 0)
    {
        std::cout << "Saving results to CSV..." << std::endl;
        std::vector<std::string> songIds;
        songIds.reserve(data.size());
        for (std::size_t i = 0; i < data.size(); ++i)
        {
            songIds.push_back(std::to_string(i));
        }

        // Create output directory and save
        try
        {
            std::string output_dir = "output";
            int result = system(("mkdir -p " + output_dir).c_str());

            if (result != 0)
            {
                std::cerr << "Warning: Could not create output directory" << std::endl;
            }

            // Save clustering results
            std::vector<SpotifyGenreRevealParty::Point> centroids(this->m_number_of_clusters);
            for (int i = 0; i < this->m_number_of_clusters; ++i)
            {
                centroids[i].clusterId = i;
                centroids[i].features.resize(this->m_number_of_dimensions);

                for (int j = 0; j < this->m_number_of_dimensions; ++j)
                {
                    centroids[i].features[j] = this->m_host_centroids[i * this->m_number_of_dimensions + j];
                }
            }

            std::string detailedFile = "output/global_gpu_results.csv";
            utils::writePointsAndCentroidsToFile(data, centroids, detailedFile);
        }
        catch (const std::exception &ex)
        {
            std::cerr << "Error creating output directory: " << ex.what() << std::endl;
        }
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_time = end_time - start_time;

    std::cout << "TIMING: GlobalGPU completed in " << elapsed_time.count() << " seconds" << std::endl;

    // Free GPU memory
    this->freeGPUMemory();
    std::cout << "Rank " << m_rank << ": GlobalGPU run complete" << std::endl;
}

void SpotifyGenreRevealParty::GlobalGPU::calculateDataDistribution(const size_t &total_points)
{
    this->m_num_points_per_process = total_points / this->m_num_processes;
    int remainder = total_points % this->m_num_points_per_process;

    // Last process gets any remainder points
    if (this->m_rank == this->m_num_processes - 1)
    {
        this->m_num_points_per_process += remainder;
    }

    if (this->m_rank == 0)
    {
        std::cout << "Data distribution: " << total_points << " total points, ";
        std::cout << "each process gets ~" << this->m_num_points_per_process << " points" << std::endl;
    }
}