#include "hip/hip_runtime.h"
//
// Created by Hunter Peart on 3/25/2025.
//

// CUDA includes at global scope
#include <hip/hip_runtime.h>

// Then include our header
#include "SharedGPU.cuh"

// Define error checking macro
#define CHECK_CUDA_ERROR(call)                                                 \
    do                                                                         \
    {                                                                          \
        hipError_t error = call;                                              \
        if (error != hipSuccess)                                              \
        {                                                                      \
            std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " \
                      << __FILE__ << ":" << __LINE__ << std::endl;             \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

// CUDA Kernel for assigning points to clusters
__global__ void assignClustersKernel(float *data, float *centroids, int *assignments, int numPoints, int numDimensions, int k)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numPoints)
    {
        float minDistance = __FLT_MAX__;
        int nearestCentroid = 0;

        // Find the nearest centroid for this data point
        for (int i = 0; i < k; ++i)
        {
            float distance = 0.0f;
            for (int j = 0; j < numDimensions; ++j)
            {
                // calculate the difference between a data point's coordinate and a centroid's coordinate along a specific dimension.
                float diff = data[idx * numDimensions + j] - centroids[i * numDimensions + j];
                distance += diff * diff;
            }

            if (distance < minDistance)
            {
                minDistance = distance;
                nearestCentroid = i;
            }
        }
        assignments[idx] = nearestCentroid;
    }
}

// CUDA Kernel for updating centroids (accumulate points)
__global__ void updateCentroidsKernel(float *data, float *centroids, int *assignments, int *clusterCounts, int numPoints, int numDimensions, int k)
{
    // Get block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numPoints)
    {
        int clusterId = assignments[idx];

        // Atomic operations to safely update shared centroids and sizes
        for (int j = 0; j < numDimensions; ++j)
        {
            atomicAdd(&centroids[clusterId * numDimensions + j], data[idx * numDimensions + j]);
        }
        atomicAdd(&clusterCounts[clusterId], 1);
    }
}

// CUDA Kernel for updating centroids (divide by cluster sizes)
__global__ void normalizeCentroidsKernel(float *centroids, int *clusterCounts, int numDimensions, int k)
{
    int clusterId = blockIdx.x * blockDim.x + threadIdx.x;

    if (clusterId < k && clusterCounts[clusterId] > 0)
    {
        for (int j = 0; j < numDimensions; ++j)
        {
            centroids[clusterId * numDimensions + j] /= clusterCounts[clusterId];
        }
    }
}

namespace SpotifyGenreRevealParty
{
    // Constructor
    SharedGPU::SharedGPU(int clusters, int maxIterations)
        : m_number_of_clusters(clusters), m_max_iterations(maxIterations),
          m_number_of_data_points(0), m_number_of_dimensions(0),
          m_device_data(nullptr), m_device_centroids(nullptr),
          m_device_cluster_assignments(nullptr), m_device_cluster_counts(nullptr)
    {
    }

    // Destructor
    SharedGPU::~SharedGPU()
    {
        freeMemory();
    }

    void SharedGPU::run(std::vector<Point> &data,
                        int k,
                        size_t dimensions,
                        int maxIterations,
                        double tolerance)
    {
        // Set the number of clusters and iterations
        m_number_of_clusters = k;
        m_max_iterations = maxIterations;

        std::vector<std::vector<float>> featureData;
        featureData.reserve(data.size());

        // Convert data to the format algorithm expects
        for (auto &&point : data)
        {
            featureData.push_back(point.features);
        }

        // Initialize GPU resources
        initialize(featureData);

        // Run KMeans on GPU
        runKMeans();

        // Update cluster assignments in the original data
        for (std::size_t i = 0; i < data.size(); ++i)
        {
            data[i].clusterId = this->m_host_cluster_assignments[i];
        }

        // Print out some info about clustering results
        std::cout << "KMeans complete with " << this->m_number_of_clusters << " clusters and " << m_max_iterations << " iterations" << std::endl;
        std::cout << "Processed " << this->m_number_of_data_points << " data points with " << m_number_of_dimensions << " dimensions" << std::endl;

        // Print cluster sizes
        std::vector<int> clusterSizes(this->m_number_of_clusters, 0);
        for (auto &&assignment : this->m_host_cluster_assignments)
        {
            clusterSizes[assignment]++;
        }

        std::cout << "Cluster sizes:" << std::endl;
        for (int i = 0; i < this->m_number_of_clusters; i++)
        {
            std::cout << "  Cluster " << i << ": " << clusterSizes[i] << " songs" << std::endl;
        }

        // Print cluster centroids
        std::cout << "Cluster centroids:" << std::endl;
        for (int i = 0; i < m_number_of_clusters; i++)
        {
            std::cout << "  Cluster " << i << ": ";
            for (int j = 0; j < m_number_of_dimensions; j++)
            {
                std::cout << m_host_centroids[i][j] << " ";
            }
            std::cout << std::endl;
        }

        // Free memory
        freeMemory();
    }

    // Initialize GPU Resources
    void SharedGPU::initialize(const std::vector<std::vector<float>> &data)
    {
        if (data.empty())
        {
            throw std::runtime_error("Empty Dataset provided");
        }

        m_number_of_data_points = data.size();
        m_number_of_dimensions = data[0].size();

        // Allocate memory on the GPU
        allocateMemory();

        // Copy data to GPU
        copyDataToDevice(data);

        // Initialize centroids (randomly select k data points as centroids)
        std::vector<int> indices(m_number_of_data_points);
        for (int i = 0; i < m_number_of_data_points; ++i)
        {
            indices[i] = i;
        }

        // Initialize random lib
        std::random_device rd;
        std::mt19937 g(rd());
        std::shuffle(indices.begin(), indices.end(), g);

        m_host_centroids.resize(m_number_of_clusters);
        for (int i = 0; i < m_number_of_clusters; ++i)
        {
            m_host_centroids[i].resize(m_number_of_dimensions);
            for (int j = 0; j < m_number_of_dimensions; ++j)
            {
                m_host_centroids[i][j] = data[indices[i]][j];
            }
        }

        // Copy centroids to GPU
        float *device_centroids = static_cast<float *>(m_device_centroids);
        for (int i = 0; i < m_number_of_clusters; ++i)
        {
            for (int j = 0; j < m_number_of_dimensions; ++j)
            {
                CHECK_CUDA_ERROR(hipMemcpy(&device_centroids[i * m_number_of_dimensions + j],
                                            &m_host_centroids[i][j],
                                            sizeof(float),
                                            hipMemcpyHostToDevice));
            }
        }
    }

    void SharedGPU::runKMeans()
    {
        const int threadPerBlock = 256;
        const int blocksForPoints = (m_number_of_data_points + threadPerBlock - 1) / threadPerBlock;
        const int blocksForCentroids = (m_number_of_clusters + threadPerBlock - 1) / threadPerBlock;

        // Cast to proper types for kernel calls
        float *device_data = static_cast<float *>(m_device_data);
        float *device_centroids = static_cast<float *>(m_device_centroids);
        int *device_cluster_assignments = static_cast<int *>(m_device_cluster_assignments);
        int *device_cluster_counts = static_cast<int *>(m_device_cluster_counts);

        // Temporary buffer for centroids
        float *d_new_centroids;
        CHECK_CUDA_ERROR(hipMalloc(&d_new_centroids, m_number_of_clusters * m_number_of_dimensions * sizeof(float)));

        // Temporary buffer for cluster counts
        for (int iter = 0; iter < m_max_iterations; ++iter)
        {
            // Step 1, assign points to clusters
            assignClustersKernel<<<blocksForPoints, threadPerBlock>>>(device_data, device_centroids, device_cluster_assignments,
                                                                      m_number_of_data_points, m_number_of_dimensions, m_number_of_clusters);
            CHECK_CUDA_ERROR(hipGetLastError());

            // Step 2, reset new centroids and cluster sizes
            CHECK_CUDA_ERROR(hipMemset(d_new_centroids, 0, m_number_of_clusters * m_number_of_dimensions * sizeof(float)));
            CHECK_CUDA_ERROR(hipMemset(device_cluster_counts, 0, m_number_of_clusters * sizeof(int)));

            // Step 3, accumulate points for each cluster
            updateCentroidsKernel<<<blocksForPoints, threadPerBlock>>>(device_data, d_new_centroids, device_cluster_assignments,
                                                                       device_cluster_counts, m_number_of_data_points,
                                                                       m_number_of_dimensions, m_number_of_clusters);
            CHECK_CUDA_ERROR(hipGetLastError());

            // Step 4. Normalize centroids
            normalizeCentroidsKernel<<<blocksForCentroids, threadPerBlock>>>(d_new_centroids, device_cluster_counts,
                                                                             m_number_of_dimensions, m_number_of_clusters);
            CHECK_CUDA_ERROR(hipGetLastError());

            // Check for convergence
            bool converged = true;
            std::vector<float> old_centroid(m_number_of_dimensions);
            std::vector<float> new_centroid(m_number_of_dimensions);

            for (int i = 0; i < m_number_of_clusters; ++i)
            {
                // Copy current centroids
                CHECK_CUDA_ERROR(hipMemcpy(old_centroid.data(),
                                            &device_centroids[i * m_number_of_dimensions],
                                            m_number_of_dimensions * sizeof(float),
                                            hipMemcpyDeviceToHost));

                // Copy new centroids
                CHECK_CUDA_ERROR(hipMemcpy(new_centroid.data(),
                                            &d_new_centroids[i * m_number_of_dimensions],
                                            m_number_of_dimensions * sizeof(float),
                                            hipMemcpyDeviceToHost));

                // Check if there's significant change
                for (int j = 0; j < m_number_of_dimensions; ++j)
                {
                    if (std::abs(old_centroid[j] - new_centroid[j]) > 1e-6)
                    {
                        converged = false;
                        break;
                    }
                }

                if (!converged)
                    break;
            }

            if (converged)
            {
                std::cout << "Converged after " << iter + 1 << " iterations" << std::endl;
                break;
            }

            // copy new centroids to centroids
            CHECK_CUDA_ERROR(hipMemcpy(device_centroids, d_new_centroids,
                                        m_number_of_clusters * m_number_of_dimensions * sizeof(float),
                                        hipMemcpyDeviceToDevice));
        }

        // Free mem buffer
        CHECK_CUDA_ERROR(hipFree(d_new_centroids));

        // Copy results back to host device
        copyResultsToHost();
    }

    std::vector<int> &SharedGPU::getClusterAssignments()
    {
        return m_host_cluster_assignments;
    }

    std::vector<std::vector<float>> &SharedGPU::getClusterCentroids()
    {
        return m_host_centroids;
    }

    // Save results to the csv
    void SharedGPU::saveResultsToCSV(const std::string &filename, const std::vector<std::string> &songIds)
    {
        // Open file
        std::ofstream file(filename);
        if (!file.is_open())
        {
            throw std::runtime_error("Failed to open file: " + filename);
        }

        // Write header
        file << "songId,cluster" << std::endl;

        // Write data
        for (std::size_t index = 0; index < songIds.size() && index < m_host_cluster_assignments.size(); ++index)
        {
            file << songIds[index] << "," << m_host_cluster_assignments[index] << std::endl;
        }

        file.close();
    }

    // Allocate memory on GPU
    void SharedGPU::allocateMemory()
    {
        // Allocate memory on the GPU
        float *device_data;
        float *device_centroids;
        int *device_cluster_assignments;
        int *device_cluster_counts;

        CHECK_CUDA_ERROR(hipMalloc(&device_data, m_number_of_data_points * m_number_of_dimensions * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&device_centroids, m_number_of_clusters * m_number_of_dimensions * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&device_cluster_assignments, m_number_of_data_points * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&device_cluster_counts, m_number_of_clusters * sizeof(int)));

        // Store as void* in the class
        m_device_data = device_data;
        m_device_centroids = device_centroids;
        m_device_cluster_assignments = device_cluster_assignments;
        m_device_cluster_counts = device_cluster_counts;

        // Resize host copies
        m_host_cluster_assignments.resize(m_number_of_data_points, 0);
    }

    // Free memory on GPU
    void SharedGPU::freeMemory()
    {
        // Free memory on the GPU
        if (m_device_data)
            CHECK_CUDA_ERROR(hipFree(m_device_data));
        if (m_device_centroids)
            CHECK_CUDA_ERROR(hipFree(m_device_centroids));
        if (m_device_cluster_assignments)
            CHECK_CUDA_ERROR(hipFree(m_device_cluster_assignments));
        if (m_device_cluster_counts)
            CHECK_CUDA_ERROR(hipFree(m_device_cluster_counts));

        // Clear host copies
        m_device_data = nullptr;
        m_device_centroids = nullptr;
        m_device_cluster_assignments = nullptr;
        m_device_cluster_counts = nullptr;
    }

    // Copy data to device
    void SharedGPU::copyDataToDevice(const std::vector<std::vector<float>> &data)
    {
        // Flatten data for more efficient copying
        std::vector<float> flatData(m_number_of_data_points * m_number_of_dimensions);
        for (int i = 0; i < m_number_of_data_points; ++i)
        {
            for (int j = 0; j < m_number_of_dimensions; ++j)
            {
                flatData[i * m_number_of_dimensions + j] = data[i][j];
            }
        }

        // Copy data to GPU
        float *device_data = static_cast<float *>(m_device_data);
        CHECK_CUDA_ERROR(hipMemcpy(device_data, flatData.data(),
                                    m_number_of_data_points * m_number_of_dimensions * sizeof(float),
                                    hipMemcpyHostToDevice));
    }

    void SharedGPU::copyResultsToHost()
    {
        // Copy cluster assignments
        int *device_cluster_assignments = static_cast<int *>(m_device_cluster_assignments);
        CHECK_CUDA_ERROR(hipMemcpy(m_host_cluster_assignments.data(), device_cluster_assignments,
                                    m_number_of_data_points * sizeof(int),
                                    hipMemcpyDeviceToHost));

        // Prepare host centroids
        m_host_centroids.resize(m_number_of_clusters, std::vector<float>(m_number_of_dimensions, 0.0f));

        // Copy centroids
        float *device_centroids = static_cast<float *>(m_device_centroids);

        // Copy centroids from device to host
        for (int i = 0; i < m_number_of_clusters; ++i)
        {
            CHECK_CUDA_ERROR(hipMemcpy(m_host_centroids[i].data(),
                                        &device_centroids[i * m_number_of_dimensions],
                                        m_number_of_dimensions * sizeof(float),
                                        hipMemcpyDeviceToHost));
        }
    }
}